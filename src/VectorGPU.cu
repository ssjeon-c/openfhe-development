//
// Created by carlosad on 2/05/24.
//
#include "VectorGPU.cuh"

namespace FIDESlib {
template <typename T>
VectorGPU<T>::VectorGPU(VectorGPU<T>&& v) noexcept
    : freeing(v.freeing), managed(v.managed), data(v.data), size(v.size), device(v.device) {
    v.freeing = true;
    v.managed = false;
}

template <typename T>
VectorGPU<T>::VectorGPU(T* data, const int size, const int device, const int offset)
    : data(data + offset), size(size), device(device), managed(false), freeing(true) {
    assert(data != nullptr);
    {
        hipPointerAttribute_t att{};
        hipPointerGetAttributes(&att, data);
        assert(att.type == hipMemoryTypeManaged || att.type == hipMemoryTypeDevice);
        assert(att.device == this->device);
    }
    CudaCheckErrorModNoSync;
    assert(size > 0);
    Out(MEMORY, "Unmanaged vector construct OK");
}

template <typename T>
VectorGPU<T>::~VectorGPU() {
    assert(freeing == true);
    Out(MEMORY, "Vector destruct OK");
}

template <typename T>
void VectorGPU<T>::free(const Stream& stream) {
    if (!managed) {
        return;
    }
    assert(!freeing);
    hipFreeAsync((void*)data, stream.ptr);
    freeing = true;
    Out(MEMORY, "Managed vector free OK");
}

template <typename T>
VectorGPU<T>::VectorGPU(const Stream& stream, const int size, const int device, const T* src)
    : data(nullptr), size(size), device(device), freeing(false), managed(true) {
    assert(size > 0);
    assert(device >= 0);
    {
        int device_count = -1;
        assert(hipGetDeviceCount(&device_count) == hipSuccess);
        assert(device < device_count);
    }
    {
        int dev = -1;
        assert(hipGetDevice(&dev) == hipSuccess);
        assert(dev == device);
        //hipSetDevice(device);
    }
    int bytes = size * sizeof(T);
    hipMallocAsync(&data, bytes, stream.ptr);
    if (src != nullptr) {
        hipMemcpyAsync(data, src, bytes, hipMemcpyHostToDevice, stream.ptr);
    }

    Out(MEMORY, "Managed vector construct OK");
}

template class VectorGPU<int>;
template class VectorGPU<void*>;
template class VectorGPU<uint32_t>;
template class VectorGPU<uint64_t>;
}  // namespace FIDESlib