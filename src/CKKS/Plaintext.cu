//
// Created by carlosad on 25/04/24.
//

#include <cereal/external/rapidjson/internal/itoa.h>
#include "CKKS/Ciphertext.cuh"
#include "CKKS/Context.cuh"
#include "CKKS/Plaintext.cuh"

namespace FIDESlib::CKKS {

Plaintext::Plaintext(Context& cc)
    : my_range(loc, LIFETIME),
      cc((CudaNvtxStart(std::string{std::source_location::current().function_name()}.substr(18 + strlen(loc))), cc)),
      c0(cc) {
    CudaNvtxStop();
}

Plaintext::Plaintext(Context& cc, const RawPlainText& raw)
    : my_range(loc, LIFETIME),
      cc((CudaNvtxStart(std::string{std::source_location::current().function_name()}.substr(18 + strlen(loc))), cc)),
      c0(cc) {
    load(raw);
    CudaNvtxStop();
}

void Plaintext::load(const RawPlainText& raw) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    c0.loadConstant(raw.sub_0, raw.moduli);
    NoiseFactor = raw.Noise;
    NoiseLevel = raw.NoiseLevel;
}

void Plaintext::store(RawPlainText& raw) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    c0.store(raw.sub_0);

    hipDeviceSynchronize();

    raw.numRes = c0.getLevel() + 1;
    raw.sub_0.resize(raw.numRes);
    c0.store(raw.sub_0);
    raw.N = cc.N;
    c0.sync();

    raw.Noise = NoiseFactor;
    raw.NoiseLevel = NoiseLevel;
    hipDeviceSynchronize();
}

void Plaintext::moddown() {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    c0.moddown(true, true);
}

bool Plaintext::adjustPlaintextToCiphertext(const Plaintext& p, const Ciphertext& c) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    if (cc.rescaleTechnique == Context::FIXEDAUTO) {
        if (p.c0.getLevel() - p.NoiseLevel > c.getLevel() - c.NoiseLevel) {
            this->copy(p);
            if (c.NoiseLevel == 1 && NoiseLevel == 2) {
                this->c0.dropToLevel(c.getLevel() + 1);
                rescale();
            } else {
                this->c0.dropToLevel(c.getLevel());
            }
            return true;
        } else if (c.NoiseLevel == 1 && p.NoiseLevel == 2) {
            this->copy(p);
            rescale();
            return true;
        } else if (p.NoiseLevel == 1 && c.NoiseLevel == 2) {
            return false;
        } else {
            this->copy(p);
            return true;
        }
    }
    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        usint c1lvl = p.c0.getLevel();
        usint c2lvl = c.getLevel();
        usint c1depth = p.NoiseLevel;
        usint c2depth = c.NoiseLevel;
        auto sizeQl1 = c1lvl + 1;
        auto sizeQl2 = c2lvl + 1;

        if (c1lvl > c2lvl) {
            this->copy(p);
            if (c1depth == 2) {
                if (c2depth == 2) {
                    double scf1 = NoiseFactor;
                    double scf2 = c.NoiseFactor;
                    double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                    double q1 =
                        cc.param.ModReduceFactor[sizeQl1 - 1];  // cryptoParams->GetModReduceFactor(sizeQl1 - 1);
                    multScalar(scf2 / scf1 * q1 / scf, false);
                    rescale();
                    if (c1lvl - 1 > c2lvl) {
                        this->c0.dropToLevel(c2lvl);
                        //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 1);
                    }
                    NoiseFactor = c.NoiseFactor;
                } else {
                    if (c1lvl - 1 == c2lvl) {
                        rescale();
                    } else {
                        double scf1 = NoiseFactor;
                        double scf2 =
                            cc.param
                                .ScalingFactorRealBig[c2lvl + 1];  //cryptoParams->GetScalingFactorRealBig(c2lvl - 1);
                        double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                        double q1 =
                            cc.param.ModReduceFactor[sizeQl1 - 1];  //cryptoParams->GetModReduceFactor(sizeQl1 - 1);
                        multScalar(scf2 / scf1 * q1 / scf, false);
                        rescale();
                        if (c1lvl - 2 > c2lvl) {
                            this->c0.dropToLevel(c2lvl + 1);
                            //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 2);
                        }
                        rescale();

                        NoiseFactor = c.NoiseFactor;
                    }
                }
            } else {
                if (c2depth == 2) {
                    double scf1 = NoiseFactor;
                    double scf2 = c.NoiseFactor;
                    double scf = cc.param.ScalingFactorReal[c1lvl];  // cryptoParams->GetScalingFactorReal(c1lvl);
                    multScalar(scf2 / scf1 / scf, false);
                    this->c0.dropToLevel(c2lvl);
                    //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl);
                    NoiseFactor = scf2;
                } else {
                    double scf1 = NoiseFactor;
                    double scf2 =
                        cc.param.ScalingFactorRealBig[c2lvl + 1];    //cryptoParams->GetScalingFactorRealBig(c2lvl - 1);
                    double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                    multScalar(scf2 / scf1 / scf, false);
                    if (c1lvl - 1 > c2lvl) {
                        this->c0.dropToLevel(c2lvl + 1);
                        //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 1);
                    }
                    rescale();
                    NoiseFactor = c.NoiseFactor;
                }
            }
            return true;
        } else if (c1lvl < c2lvl) {
            return false;
        } else {
            this->copy(p);
            if (c1depth < c2depth) {
                multScalar(1.0, false);
            } else if (c2depth < c1depth) {
                rescale();
            }
            return true;
        }
    }
    return false;
}
void Plaintext::copy(const Plaintext& p) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    this->c0.copy(p.c0);
    this->NoiseFactor = p.NoiseFactor;
    this->NoiseLevel = p.NoiseLevel;
}
void Plaintext::multScalar(double c, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    /*
    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT ||
        cc.rescaleTechnique == Context::FIXEDAUTO) {
        if (NoiseLevel == 2)
            this->rescale();
    }
    assert(this->NoiseLevel == 1);
    */
    auto elem = cc.ElemForEvalMult(c0.getLevel(), c);
    c0.multScalar(elem);

    if (rescale) {
        c0.rescale();
    }
    // Manage metadata
    NoiseLevel += 1;
    NoiseFactor *= cc.param.ScalingFactorReal.at(c0.getLevel() + rescale);
    if (rescale) {
        NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + rescale);
        NoiseLevel -= 1;
    }
}
void Plaintext::rescale() {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    assert(this->NoiseLevel >= 2);

    c0.rescale();

    // Manage metadata
    NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
    NoiseLevel -= 1;
}

}  // namespace FIDESlib::CKKS