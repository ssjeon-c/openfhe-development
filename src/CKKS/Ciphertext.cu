#include "hip/hip_runtime.h"
//
// Created by carlosad on 24/04/24.
//

#include "CKKS/Ciphertext.cuh"
#include "CKKS/Context.cuh"
#include "CKKS/KeySwitchingKey.cuh"
#include "CKKS/Plaintext.cuh"

namespace FIDESlib::CKKS {

Ciphertext::Ciphertext(Context& cc)
    : my_range(loc, LIFETIME),
      cc((CudaNvtxStart(std::string{std::source_location::current().function_name()}.substr(18 + strlen(loc))), cc)),
      c0(cc),
      c1(cc) {
    CudaNvtxStop();
}

Ciphertext::Ciphertext(Context& cc, const RawCipherText& rawct)
    : my_range(loc, LIFETIME),
      cc((CudaNvtxStart(std::string{std::source_location::current().function_name()}.substr(18 + strlen(loc))), cc)),
      c0(cc, rawct.sub_0),
      c1(cc, rawct.sub_1) {
    NoiseLevel = rawct.NoiseLevel;
    NoiseFactor = rawct.Noise;
    CudaNvtxStop();
}

void Ciphertext::add(const Ciphertext& b) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (cc.rescaleTechnique == Context::FIXEDAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTO ||
        cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        if (!adjustForAddOrSub(b)) {
            Ciphertext b_(cc);
            b_.copy(b);
            if (b_.adjustForAddOrSub(*this))
                add(b_);
            else
                assert(false);
            return;
        }
    }

    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        assert(this->getLevel() == b.getLevel());
    } else if (getLevel() > b.getLevel()) {
        c0.dropToLevel(b.getLevel());
        c1.dropToLevel(b.getLevel());
    }

    c0.add(b.c0);
    c1.add(b.c1);
}

void Ciphertext::sub(const Ciphertext& b) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (cc.rescaleTechnique == Context::FIXEDAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTO ||
        cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        if (!adjustForAddOrSub(b)) {
            Ciphertext b_(cc);
            b_.copy(b);
            if (b_.adjustForAddOrSub(*this))
                sub(b_);
            else
                assert(false);
            return;
        }
    }

    c0.sub(b.c0);
    c1.sub(b.c1);
}

void Ciphertext::addPt(const Plaintext& b) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT ||
        cc.rescaleTechnique == Context::FIXEDAUTO) {
        if (b.c0.getLevel() != this->getLevel() ||
            (b.NoiseLevel == 1 && NoiseLevel == 2) /*!hasSameScalingFactor(b)*/) {
            Plaintext b_(cc);
            if (!b_.adjustPlaintextToCiphertext(b, *this)) {
                assert(false);
            } else {
                addPt(b_);
            }
            return;
        }
    }
    assert(NoiseLevel == b.NoiseLevel);

    c0.add(b.c0);
    //NoiseFactor += b.NoiseFactor;
}

void Ciphertext::load(const RawCipherText& rawct) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    c0.load(rawct.sub_0, rawct.moduli);
    c1.load(rawct.sub_1, rawct.moduli);

    NoiseLevel = rawct.NoiseLevel;
    NoiseFactor = rawct.Noise;
}

void Ciphertext::store(const Context& cc, RawCipherText& rawct) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    hipDeviceSynchronize();
    rawct.numRes = c0.getLevel() + 1;
    rawct.sub_0.resize(rawct.numRes);
    rawct.sub_1.resize(rawct.numRes);
    c0.store(rawct.sub_0);
    c1.store(rawct.sub_1);
    rawct.N = cc.N;
    c0.sync();
    c1.sync();

    rawct.NoiseLevel = NoiseLevel;
    rawct.Noise = NoiseFactor;
    hipDeviceSynchronize();
}

void Ciphertext::modDown() {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    c0.moddown(true, false);
    c1.moddown(true, false);
    c0.freeSpecialLimbs();
    c1.freeSpecialLimbs();
}

void Ciphertext::modUp() {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    c0.modup();
    //c1.modup();
}

void Ciphertext::multPt(const Plaintext& b, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (cc.rescaleTechnique == Context::FIXEDAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTO ||
        cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        if (NoiseLevel == 2)
            this->rescale();
    }

    if (cc.rescaleTechnique == Context::FIXEDAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTO ||
        cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        if (b.c0.getLevel() != this->getLevel() || b.NoiseLevel == 2 /*!hasSameScalingFactor(b)*/) {
            Plaintext b_(cc);
            if (!b_.adjustPlaintextToCiphertext(b, *this)) {
                assert(false);
            } else {
                if (NoiseLevel == 2)
                    this->rescale();
                if (b_.NoiseLevel == 2)
                    b_.rescale();
                multPt(b_, rescale);
            }
            return;
        }
    }

    assert(NoiseLevel < 2);
    assert(b.NoiseLevel < 2);
    c0.multPt(b.c0, rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL);
    c1.multPt(b.c0, rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL);

    // Manage metadata
    NoiseLevel += b.NoiseLevel;
    NoiseFactor *= b.NoiseFactor;
    if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL) {
        NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
        NoiseLevel -= 1;
    }
}

void Ciphertext::rescale() {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    assert(this->NoiseLevel == 2);
    if (cc.rescaleTechnique != Context::FIXEDMANUAL) {
        // this wouldn't do anything in OpenFHE
    }

    c0.rescale();
    c1.rescale();

    // Manage metadata
    NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
    NoiseLevel -= 1;
}

void Ciphertext::mult(const Ciphertext& b, const KeySwitchingKey& kskEval, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (cc.rescaleTechnique == Context::FIXEDAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTO ||
        cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        if (!adjustForMult(b)) {
            Ciphertext b_(cc);
            b_.copy(b);
            if (b_.adjustForMult(*this))
                mult(b_, kskEval, rescale);
            else
                assert(false);
            return;
        }
    }
    assert(NoiseLevel == 1);
    assert(NoiseLevel == b.NoiseLevel);
    /*
    if (getLevel() > b.getLevel()) {
        this->c0.dropToLevel(b.getLevel());
        this->c1.dropToLevel(b.getLevel());
    }
    */
    assert(c0.getLevel() <= b.c0.getLevel());
    assert(c1.getLevel() <= b.c1.getLevel());
    constexpr bool PRINT = false;
    Out(KEYSWITCH, " start ");

    if constexpr (0) {
        cc.getKeySwitchAux().setLevel(c1.getLevel());
        cc.getKeySwitchAux().multElement(c1, b.c1);
        cc.getKeySwitchAux().modup();

        auto& aux0 = cc.getKeySwitchAux().dotKSKInPlace(kskEval, c0.getLevel());

        hipDeviceSynchronize();
        /*
        std::vector<uint64_t> p(c1.getLevel() + 1);
        for (int i = 0; i <= c1.getLevel(); ++i)
            p[i] = hC_.P[i];

        cc.getKeySwitchAux().addScalar(p);
        */
        c1.mult1AddMult23Add4(b.c0, c0, b.c1, cc.getKeySwitchAux());  // Read 4 first for better cache locality.
        hipDeviceSynchronize();
        cc.getKeySwitchAux().moddown(true, false);
        hipDeviceSynchronize();
        c1.copy(cc.getKeySwitchAux());
        hipDeviceSynchronize();
        /*
        for (int i = 0; i <= c1.getLevel(); ++i)
            p[i] = 1.0;
        cc.getKeySwitchAux().subScalar(p);
        */
        hipDeviceSynchronize();
        c0.mult1Add2(b.c0, aux0);
        aux0.moddown(true, false);
        c0.copy(aux0);
        hipDeviceSynchronize();
        //c1.mult1AddMult23Add4(b.c0, c0, b.c1, cc.getKeySwitchAux());  // Read 4 first for better cache locality.

        if (rescale) {
            c1.rescale();
        }
        if (rescale) {
            c0.rescale();
        }
    } else {
        cc.getKeySwitchAux().multModupDotKSK(c1, b.c1, c0, b.c0, kskEval);
        c1.moddown(true, false);
        if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL)
            c1.rescale();
        c0.moddown(true, false);
        if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL)
            c0.rescale();
    }

    // Manage metadata
    NoiseLevel += b.NoiseLevel;
    NoiseFactor *= b.NoiseFactor;
    if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL) {
        NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
        NoiseLevel -= 1;
    }
    Out(KEYSWITCH, " finish ");
}

void Ciphertext::square(const KeySwitchingKey& kskEval, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    constexpr bool PRINT = false;
    Out(KEYSWITCH, " start ");

    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT ||
        cc.rescaleTechnique == Context::FIXEDAUTO) {
        if (NoiseLevel == 2)
            this->rescale();
    }
    assert(this->NoiseLevel == 1);

    if constexpr (0) {
        cc.getKeySwitchAux().setLevel(c1.getLevel());
        cc.getKeySwitchAux().squareElement(c1);
        cc.getKeySwitchAux().modup();
        auto& aux0 = cc.getKeySwitchAux().dotKSKInPlace(kskEval, c0.getLevel());
        cc.getKeySwitchAux().moddown();
        aux0.moddown(true, false);
        //c1.mult1AddMult23Add4(c0, c0, c1, cc.getKeySwitchAux());
        c1.binomialSquareFold(c0, aux0, cc.getKeySwitchAux());
        if (rescale) {
            c1.rescale();
            c0.rescale();
        }
        //   //
        // Manage metadata
        NoiseLevel += NoiseLevel;
        NoiseFactor *= NoiseFactor;
        if (rescale) {
            NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
            NoiseLevel -= 1;
        }
    } else if constexpr (1) {
        cc.getKeySwitchAux().squareModupDotKSK(c0, c1, kskEval);

        c1.moddown(true, false);
        if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL)
            c1.rescale();
        c0.moddown(true, false);
        if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL)
            c0.rescale();

        NoiseLevel += NoiseLevel;
        NoiseFactor *= NoiseFactor;
        if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL) {
            NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
            NoiseLevel -= 1;
        }
    } else {
        this->mult(*this, kskEval, rescale);
    }
    Out(KEYSWITCH, " finish ");
}

void Ciphertext::multScalarNoPrecheck(const double c, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    auto elem = cc.ElemForEvalMult(c0.getLevel(), c);
    c0.multScalar(elem);
    c1.multScalar(elem);

    // Manage metadata
    NoiseLevel += 1;
    NoiseFactor *= cc.param.ScalingFactorReal.at(c0.getLevel());
    if (rescale) {
        NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel());
        NoiseLevel -= 1;
    }

    if (rescale) {
        c0.rescale();
        c1.rescale();
    }
}

void Ciphertext::multScalar(const double c, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT ||
        cc.rescaleTechnique == Context::FIXEDAUTO) {
        if (NoiseLevel == 2)
            this->rescale();
    }
    assert(this->NoiseLevel == 1);
    multScalarNoPrecheck(c, rescale && cc.rescaleTechnique == Context::FIXEDMANUAL);
}

void Ciphertext::addScalar(const double c) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    auto elem = cc.ElemForEvalAddOrSub(c0.getLevel(), std::abs(c), this->NoiseLevel);

    if (c >= 0.0) {
        c0.addScalar(elem);
    } else {
        c0.subScalar(elem);
    }
}

void Ciphertext::automorph(const int index, const int br) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    c0.automorph(index, br);
    c1.automorph(index, br);
}

void Ciphertext::automorph_multi(const int index, const int br) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    c0.automorph_multi(index, br);
    c1.automorph_multi(index, br);
}

void Ciphertext::rotate(const int index, const KeySwitchingKey& kskRot) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    constexpr bool PRINT = false;

    if constexpr (0) {
        if constexpr (PRINT) {
            std::cout << "Output Automorph 1.";
            for (auto& j : c1.GPU)
                for (auto& i : j.limb) {
                    SWITCH(i, printThisLimb(2));
                }
        }
        c1.modupInto(cc.getKeySwitchAux());
        RNSPoly& aux0 = c1.dotKSKInPlaceFrom(cc.getKeySwitchAux(), kskRot, c1.getLevel());
        c1.moddown();
        if constexpr (PRINT) {
            std::cout << "Output Automorph 1.";
            for (auto& j : c1.GPU)
                for (auto& i : j.limb) {
                    SWITCH(i, printThisLimb(2));
                }
        }
        c1.automorph(index, 1);

        aux0.moddown(true, false);
        if constexpr (PRINT) {
            std::cout << "c0\n";
            for (auto& j : c0.GPU)
                for (auto& i : j.limb) {
                    SWITCH(i, printThisLimb(2));
                }
        }
        c0.add(aux0);
        if constexpr (PRINT) {
            std::cout << "Output KeySwitch 0.";
            for (auto& j : aux0.GPU)
                for (auto& i : j.limb) {
                    SWITCH(i, printThisLimb(2));
                }
        }

        if constexpr (PRINT) {
            std::cout << "Output Add 0.";
            for (auto& j : c0.GPU)
                for (auto& i : j.limb) {
                    SWITCH(i, printThisLimb(2));
                }
        }
        c0.automorph(index, 1);
        if constexpr (PRINT) {
            std::cout << "Output Rot 0.";
            for (auto& j : c0.GPU)
                for (auto& i : j.limb) {
                    SWITCH(i, printThisLimb(2));
                }
        }
    } else if constexpr (1) {

        cc.getKeySwitchAux().rotateModupDotKSK(c0, c1, kskRot);

        c1.moddown(true, false);
        c1.automorph(index, 1);
        c0.moddown(true, false);
        c0.automorph(index, 1);
    } else {

        c1.modupInto(cc.getKeySwitchAux());
        RNSPoly& aux0 = c1.dotKSKInPlaceFrom(cc.getKeySwitchAux(), kskRot, c1.getLevel());
        c1.moddown();
        c1.automorph(index, 1);

        aux0.moddown(true, false);
        c0.add(aux0);
        c0.automorph(index, 1);
    }
}

void Ciphertext::rotate(const Ciphertext& c, const int index, const KeySwitchingKey& kskRot) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    this->copy(c);
    this->rotate(index, kskRot);
}

void Ciphertext::conjugate(const Ciphertext& c) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    this->copy(c);
    //this->rotate(2 * cc.N - 1, cc.GetRotationKey(2 * cc.N - 1));

    int index = 2 * cc.N - 1;
    cc.getKeySwitchAux().setLevel(c1.getLevel());
    c1.modupInto(cc.getKeySwitchAux());
    RNSPoly& aux0 = c1.dotKSKInPlaceFrom(cc.getKeySwitchAux(), cc.GetRotationKey(index), c1.getLevel());
    c1.moddown(true, true);
    //c1.automorph(index, 1);

    for (int i = 0; i < (int)c1.GPU.size(); ++i) {
        c1.GPU.at(i).automorph(index, 1);
    }
    aux0.moddown(true, false);
    c0.add(aux0);
    //c0.automorph(index, 1);
    /*
    for (auto& i : cc.GetRotationKey(index).a.GPU) {
        for (auto& j : i.DIGITlimb) {
            for (auto& k : j) {
                SWITCH(k, printThisLimb(1));
            }
        }
    }
    std::cout << std::endl;
    for (auto& i : cc.GetRotationKey(index).b.GPU) {
        for (auto& j : i.DIGITlimb) {
            for (auto& k : j) {
                SWITCH(k, printThisLimb(1));
            }
        }
    }
    std::cout << std::endl;
*/
    for (int i = 0; i < (int)c0.GPU.size(); ++i) {
        c0.GPU.at(i).automorph(index, 1);
    }
}

void Ciphertext::rotate_hoisted(const std::vector<KeySwitchingKey*>& ksk, const std::vector<int>& indexes,
                                std::vector<Ciphertext*> results) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    constexpr bool PRINT = 0;
    assert(ksk.size() == results.size());
    for (auto& i : results) {
        if (this->c0.getLevel() > i->c0.getLevel()) {
            if (i->c0.getLevel() == -1) {
                i->c0.grow(this->c0.getLevel(), true);
            } else {
                assert("Ciphertext initialized but to the wrong level" == nullptr);
            }
        }
        if (this->c1.getLevel() > i->c1.getLevel()) {
            if (i->c1.getLevel() == -1) {
                i->c1.grow(this->c1.getLevel(), true);
            } else {
                assert("Ciphertext initialized but to the wrong level" == nullptr);
            }
        }
    }
    c1.modupInto(cc.getKeySwitchAux());

    for (int i = 0; i < ksk.size(); ++i) {
        if (indexes[i] == 0) {
            results[i]->copy(*this);
        } else {
            RNSPoly& aux0 = results[i]->c1.dotKSKInPlaceFrom(cc.getKeySwitchAux(), *ksk[i], c1.getLevel(), &c1);
            results[i]->c0.dropToLevel(getLevel());
            results[i]->c1.dropToLevel(getLevel());

            results[i]->c1.moddown();
            results[i]->c1.automorph(indexes[i], 1);
            aux0.moddown(true, false);
            results[i]->c0.add(c0, aux0);
            results[i]->c0.automorph(indexes[i], 1);
            results[i]->NoiseLevel = NoiseLevel;
            results[i]->NoiseFactor = NoiseFactor;
        }
    }
}
void Ciphertext::mult(const Ciphertext& b, const Ciphertext& c, const KeySwitchingKey& kskEval, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (this == &b && this == &c) {
        this->square(kskEval, rescale);
    } else if (this == &b) {
        this->mult(c, kskEval, rescale);
    } else if (this == &c) {
        this->mult(b, kskEval, rescale);
    } else {
        if (b.getLevel() <= c.getLevel()) {
            this->copy(b);
            this->mult(c, kskEval, rescale);
        } else {
            this->copy(c);
            this->mult(b, kskEval, rescale);
        }
    }
}

void Ciphertext::square(const Ciphertext& src, const KeySwitchingKey& kskEval, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (this == &src) {
        this->square(kskEval, rescale);
    } else {
        this->copy(src);
        this->square(kskEval, rescale);
    }
}
void Ciphertext::dropToLevel(int level) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    c0.dropToLevel(level);
    c1.dropToLevel(level);
}
int Ciphertext::getLevel() const {
    assert(c0.getLevel() == c1.getLevel());
    return c0.getLevel();
}
void Ciphertext::multScalar(const Ciphertext& b, const double c, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    this->copy(b);
    this->multScalar(c, rescale);
}
void Ciphertext::evalLinearWSumMutable(uint32_t n, const std::vector<Ciphertext>& ctxs, std::vector<double> weights) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if constexpr (1) {
        this->c0.grow(ctxs[0].getLevel(), true);
        this->c1.grow(ctxs[0].getLevel(), true);
        this->NoiseLevel = 1;

        for (int i = 0; i < n; ++i) {
            if (cc.rescaleTechnique == Context::FIXEDMANUAL) {
                assert(ctxs[i].NoiseLevel == 1);
                assert(getLevel() <= ctxs[i].getLevel());
            } else {
                assert(ctxs[i].NoiseLevel == 1);
                assert(getLevel() == ctxs[i].getLevel());
            }
        }

        std::vector<uint64_t> elem;
        for (int i = 0; i < n; ++i) {
            auto aux = cc.ElemForEvalMult(c0.getLevel(), weights[i]);
            for (auto j : aux)
                elem.push_back(j);
        }

        std::vector<const RNSPoly*> c0s(n), c1s(n);

        for (int i = 0; i < n; ++i) {
            c0s[i] = &ctxs[i].c0;
            c1s[i] = &ctxs[i].c1;
        }
        c0.evalLinearWSum(n, c0s, elem);
        c1.evalLinearWSum(n, c1s, elem);

        this->NoiseLevel = 2;
        this->NoiseFactor = ctxs[0].NoiseFactor;
        NoiseFactor *= cc.param.ScalingFactorReal.at(c0.getLevel());
    } else {
        this->multScalar(ctxs[0], weights[0], false);
        for (int i = 1; i < n; ++i) {
            assert(getLevel() <= ctxs[i].getLevel());
        }
        for (int i = 1; i < n; ++i) {
            this->addMultScalar(ctxs[i], weights[i]);
        }
    }
}
void Ciphertext::addMultScalar(const Ciphertext& b, double d) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    assert(NoiseLevel == 2);
    assert(b.NoiseLevel == 1);
    assert(b.getLevel() >= getLevel());
    auto elem = cc.ElemForEvalMult(c0.getLevel(), d);

    RNSPoly aux0(cc);
    RNSPoly aux1(cc);
    aux0.copy(b.c0);
    aux0.multScalar(elem);
    c0.add(aux0);
    aux1.copy(b.c1);
    aux1.multScalar(elem);
    c1.add(aux1);
}
void Ciphertext::addScalar(const Ciphertext& b, double c) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    this->copy(b);
    this->addScalar(c);
}
void Ciphertext::add(const Ciphertext& b, const Ciphertext& c) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    assert(NoiseLevel <= 2);
    if (this == &b && this == &c) {
        this->add(c);  // TODO improve for less memory reads
    } else if (this == &b) {
        this->add(c);
    } else if (this == &c) {
        this->add(b);
    } else {
        if (b.getLevel() <= c.getLevel()) {
            this->copy(b);
            this->add(c);
        } else {
            this->copy(c);
            this->add(b);
        }
    }
}
void Ciphertext::copy(const Ciphertext& ciphertext) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    c0.copy(ciphertext.c0);
    c1.copy(ciphertext.c1);
    //hipDeviceSynchronize();
    this->NoiseLevel = ciphertext.NoiseLevel;
    this->NoiseFactor = ciphertext.NoiseFactor;
}
void Ciphertext::multPt(const Ciphertext& c, const Plaintext& b, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    this->copy(c);
    multPt(b, rescale);
}
void Ciphertext::addMultPt(const Ciphertext& c, const Plaintext& b, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    assert(NoiseLevel == 2);
    assert(c.NoiseLevel == 1);
    assert(b.NoiseLevel == 1);

    c0.addMult(c.c0, b.c0);
    c1.addMult(c.c1, b.c0);

    if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL) {
        c0.rescale();
        c1.rescale();
    }

    //NoiseFactor += c.NoiseFactor * b.NoiseFactor;
    if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL) {
        NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
        NoiseLevel -= 1;
    }
}
void Ciphertext::addPt(const Ciphertext& ciphertext, const Plaintext& plaintext) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    this->copy(ciphertext);
    this->addPt(plaintext);
}

void Ciphertext::sub(const Ciphertext& ciphertext, const Ciphertext& ciphertext1) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    assert(ciphertext.getLevel() <= ciphertext1.getLevel());
    this->copy(ciphertext);
    this->sub(ciphertext1);
}
bool Ciphertext::adjustForAddOrSub(const Ciphertext& b) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (cc.rescaleTechnique == Context::FIXEDMANUAL) {
        if (b.NoiseLevel > NoiseLevel || (b.getLevel() < getLevel()))
            return false;
        else
            return true;
    } else if (cc.rescaleTechnique == Context::FIXEDAUTO) {
        if (getLevel() - NoiseLevel > b.getLevel() - b.NoiseLevel) {
            if (b.NoiseLevel == 1 && NoiseLevel == 2) {
                this->dropToLevel(b.getLevel() + 1);
                rescale();
            } else {
                this->dropToLevel(b.getLevel());
            }
            return true;
        } else if (b.NoiseLevel == 1 && NoiseLevel == 2) {
            rescale();
            return true;
        } else if (NoiseLevel == 1 && b.NoiseLevel == 2) {
            return false;
        } else {
            return true;
        }
    } else if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        usint c1lvl = getLevel();
        usint c2lvl = b.getLevel();
        usint c1depth = this->NoiseLevel;
        usint c2depth = b.NoiseLevel;
        auto sizeQl1 = c1lvl + 1;
        auto sizeQl2 = c2lvl + 1;

        if (c1lvl > c2lvl) {
            if (c1depth == 2) {
                if (c2depth == 2) {
                    double scf1 = NoiseFactor;
                    double scf2 = b.NoiseFactor;
                    double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                    double q1 =
                        cc.param.ModReduceFactor[sizeQl1 - 1];  // cryptoParams->GetModReduceFactor(sizeQl1 - 1);
                    multScalarNoPrecheck(scf2 / scf1 * q1 / scf, true);
                    if (c1lvl - 1 > c2lvl) {
                        this->dropToLevel(c2lvl);
                        //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 1);
                    }
                    NoiseFactor = b.NoiseFactor;
                } else {
                    if (c1lvl - 1 == c2lvl) {
                        rescale();
                    } else {
                        double scf1 = NoiseFactor;
                        double scf2 =
                            cc.param
                                .ScalingFactorRealBig[c2lvl + 1];  //cryptoParams->GetScalingFactorRealBig(c2lvl - 1);
                        double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                        double q1 =
                            cc.param.ModReduceFactor[sizeQl1 - 1];  //cryptoParams->GetModReduceFactor(sizeQl1 - 1);
                        multScalarNoPrecheck(scf2 / scf1 * q1 / scf, true);
                        if (c1lvl - 2 > c2lvl) {
                            this->dropToLevel(c2lvl + 1);
                            //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 2);
                        }
                        rescale();

                        NoiseFactor = b.NoiseFactor;
                    }
                }
            } else {
                if (c2depth == 2) {
                    double scf1 = NoiseFactor;
                    double scf2 = b.NoiseFactor;
                    double scf = cc.param.ScalingFactorReal[c1lvl];  // cryptoParams->GetScalingFactorReal(c1lvl);
                    multScalarNoPrecheck(scf2 / scf1 / scf);
                    this->dropToLevel(c2lvl);
                    //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl);
                    NoiseFactor = scf2;
                } else {
                    double scf1 = NoiseFactor;
                    double scf2 =
                        cc.param.ScalingFactorRealBig[c2lvl + 1];    //cryptoParams->GetScalingFactorRealBig(c2lvl - 1);
                    double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                    multScalarNoPrecheck(scf2 / scf1 / scf);
                    if (c1lvl - 1 > c2lvl) {
                        this->dropToLevel(c2lvl + 1);
                        //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 1);
                    }
                    rescale();
                    NoiseFactor = b.NoiseFactor;
                }
            }
            return true;
        } else if (c1lvl < c2lvl) {
            return false;
        } else {
            if (c1depth < c2depth) {
                multScalar(1.0, false);
            } else if (c2depth < c1depth) {
                return false;
            }
            return true;
        }
    }
}

bool Ciphertext::adjustForMult(const Ciphertext& ciphertext) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (adjustForAddOrSub(ciphertext)) {
        if (NoiseLevel == 2)
            rescale();
        if (ciphertext.NoiseLevel == 2)
            return false;
        else
            return true;
    } else {
        if (NoiseLevel == 2)
            rescale();
        return false;
    }
}
bool Ciphertext::hasSameScalingFactor(const Plaintext& b) const {
    return NoiseFactor > b.NoiseFactor * (1 - 1e-9) && NoiseFactor < b.NoiseFactor * (1 + 1e-9);
}

}  // namespace FIDESlib::CKKS
