#include "hip/hip_runtime.h"
//
// Created by carlosad on 2/05/24.
//
#include "CKKS/Context.cuh"

#include <source_location>
#include "CKKS/BootstrapPrecomputation.cuh"
#include "CKKS/KeySwitchingKey.cuh"

namespace FIDESlib::CKKS {

std::vector<dim3> Context::generateLimbGPUid(const std::vector<std::vector<LimbRecord>>& meta, const int L) {
    std::vector<dim3> res(L + 1, 0);
    for (int i = 0; i < static_cast<int>(meta.size()); ++i) {
        for (size_t j = 0; j < meta.at(i).size(); ++j) {
            res.at(meta[i][j].id) = {static_cast<uint32_t>(i), static_cast<uint32_t>(j)};
        }
    }
    return res;
}

std::vector<std::vector<std::vector<LimbRecord>>> Context::generateDigitMeta(
    const std::vector<std::vector<LimbRecord>>& meta, const std::vector<LimbRecord>& specialMeta,
    const std::vector<std::vector<int>> digitGPUid) {
    std::vector<std::vector<std::vector<LimbRecord>>> digitMeta(meta.size());

    for (size_t i = 0; i < digitGPUid.size(); ++i) {
        for (int d : digitGPUid.at(i)) {
            digitMeta[i].emplace_back();
            for (auto& l : specialMeta) {
                digitMeta[i].back().emplace_back(LimbRecord{.id = l.id, .type = l.type, .digit = l.digit});
                digitMeta[i].back().back().stream.init();
            }

            for (auto& l : meta.at(i)) {
                if (l.digit != d) {
                    digitMeta[i].back().emplace_back(LimbRecord{.id = l.id, .type = l.type, .digit = l.digit});
                    digitMeta[i].back().back().stream.init();
                }
            }
        }
    }
    return digitMeta;
}

std::vector<std::vector<std::vector<LimbRecord>>> Context::generateDecompMeta(
    const std::vector<std::vector<LimbRecord>>& meta, const std::vector<std::vector<int>> digitGPUid) {
    std::vector<std::vector<std::vector<LimbRecord>>> decompMeta(meta.size());

    for (size_t i = 0; i < digitGPUid.size(); ++i) {
        for (int d : digitGPUid.at(i)) {
            decompMeta[i].emplace_back();
            for (auto& l : meta.at(i)) {
                if (l.digit == d) {
                    decompMeta[i].back().push_back(LimbRecord{.id = l.id, .type = l.type, .digit = l.digit});
                    decompMeta[i].back().back().stream.init();
                }
            }
        }
    }

    return decompMeta;
}

bool Context::isValidPrimeId(const int i) const {
    return (i >= 0 && i < L + 1 + K);
}

Context::Context(Parameters param, const std::vector<int>& devs, const int secBits)
    : my_range(loc, LIFETIME),
      param((CudaNvtxStart(std::string{std::source_location::current().function_name()}.substr(18 + strlen(loc))),
             param)),
      logN(param.logN),
      N(1 << logN),
      slots(1 << (logN - 1)),
      rescaleTechnique(translateRescalingTechnique(param.scalingTechnique)),
      L(param.L),
      logQ(computeLogQ(L, param.primes)),
      batch(param.batch),
      GPUid(devs),
      dnum(validateDnum(GPUid, param.dnum)),
      digitGPUid(generateDigitGPUid(dnum, devs)),
      prime(param.primes.begin(), param.primes.begin() + L + 1),
      meta{generateMeta(GPUid, dnum, digitGPUid, prime, param)},
      logQ_d(computeLogQ_d(dnum, meta, prime)),
      K(computeK(logQ_d, param.Sprimes, param)),
      logP(computeLogQ(K - 1, param.Sprimes)),
      specialPrime(param.Sprimes.begin(), param.Sprimes.begin() + K),
      specialMeta(generateSpecialMeta(meta, specialPrime, L + 1)),
      decompMeta(generateDecompMeta(meta, digitGPUid)),
      digitMeta(generateDigitMeta(meta, specialMeta, digitGPUid)),
      limbGPUid(generateLimbGPUid(meta, L)),
      GPUrank(GPUid.size(), -1) {

    SetupConstants<Parameters>(prime, meta, specialPrime, specialMeta, decompMeta, digitMeta, digitGPUid, GPUid, N,
                               param);
    // PrepareNCCLCommunication();
    // CheckBitSecurity();
    for (int dev : GPUid) {
        hipMemPool_t mp;
        hipDeviceGetDefaultMemPool(&mp, dev);
        uint64_t threshold = UINT64_MAX;  //5l * 1024l * 1024l * 1024l;  // One Gigabyte of memory
        hipMemPoolSetAttribute(mp, hipMemPoolAttrReleaseThreshold, &threshold);
        CudaCheckErrorModNoSync;
    }

    CudaNvtxStop();
}

int Context::computeLogQ(const int L, std::vector<PrimeRecord>& primes) {
    int res = 0;
    assert(L <= (int)primes.size());
    for (int i = 0; i <= L; ++i) {
        res += (primes[i].bits == -1) ? (primes[i].bits = (int)std::bit_width(primes[i].p)) : primes[i].bits;
    }
    return res;
}

int Context::validateDnum(const std::vector<int>& GPUid, const int dnum) {
    return dnum;
}

int findDigitOnParam(const Parameters& param, uint64_t modulus) {
    for (size_t i = 0; i < param.raw->PARTITIONmoduli.size(); ++i) {
        for (uint64_t j : param.raw->PARTITIONmoduli.at(i)) {
            if (modulus == j)
                return i;
        }
    }
    return -1;
}

std::vector<std::vector<LimbRecord>> Context::generateMeta(const std::vector<int>& GPUid, const int dnum,
                                                           const std::vector<std::vector<int>> digitGPUid,
                                                           const std::vector<PrimeRecord>& prime,
                                                           const Parameters& param) {
    int devs = GPUid.size();
    std::vector<std::vector<LimbRecord>> meta(devs);

    //for (int i = 0; i < devs; ++i) {
    // hipSetDevice(GPUid.at(i));
    // meta.at(i).resize((prime.size() + devs - i - 1) / devs);
    //}

    for (int i = 0; i < (int)prime.size(); ++i) {
        int digit = param.raw == nullptr ? i % dnum : findDigitOnParam(param, prime.at(i).p);

        int dev = -1;
        for (size_t j = 0; j < digitGPUid.size(); ++j) {
            for (auto& k : digitGPUid.at(j))
                if (k == digit)
                    dev = j;
        }

        meta[dev].push_back(LimbRecord{
            .id = i, .type = (prime[i].type ? *(prime[i].type) : (prime[i].bits <= 30 ? U32 : U64)), .digit = digit});
        meta[dev].back().stream.init();
    }

    return meta;
}

std::vector<int> Context::computeLogQ_d(const int dnum, const std::vector<std::vector<LimbRecord>>& meta,
                                        const std::vector<PrimeRecord>& prime) {
    std::vector<int> logQ_d(dnum, 0);

    for (auto& i : meta)
        for (auto& j : i)
            logQ_d.at(j.digit) += prime.at(j.id).bits;

    return logQ_d;
}

int Context::computeK(const std::vector<int>& logQ_d, std::vector<PrimeRecord>& Sprimes, const Parameters& param) {

    size_t res = 0;
    int logMaxD = *std::max_element(logQ_d.begin(), logQ_d.end());
    int bits = 0;
    for (; bits < logMaxD && res < Sprimes.size(); ++res) {
        bits += (Sprimes.at(res).bits <= 0) ? (Sprimes.at(res).bits = (int)std::bit_width(Sprimes.at(res).p)) - 1
                                            : Sprimes.at(res).bits - 1;
    }

    if (param.K != -1) {
        return param.K;
    }
    assert(bits >= logMaxD);
    return res;
}

std::vector<LimbRecord> Context::generateSpecialMeta(const std::vector<std::vector<LimbRecord>>& meta,
                                                     const std::vector<PrimeRecord>& specialPrime, const int ID0) {
    std::vector<LimbRecord> specialMeta(specialPrime.size());

    for (int i = 0; i < (int)specialPrime.size(); ++i) {
        specialMeta.at(i).id = ID0 + i;
        specialMeta.at(i).type =
            (specialPrime[i].type ? *(specialPrime[i].type) : (specialPrime[i].bits <= 30 ? U32 : U64));
        specialMeta.at(i).stream.init();
    }

    return specialMeta;
}

std::vector<std::vector<int>> Context::generateDigitGPUid(const int dnum, const std::vector<int>& devs) {
    std::vector<std::vector<int>> res(devs.size());
    for (int d = 0; d < dnum; ++d) {
        res[d % res.size()].push_back(d);
    }
    return res;
}
RNSPoly& Context::getKeySwitchAux() {
    if (key_switch_aux == nullptr)
        key_switch_aux = std::make_unique<RNSPoly>(*this, L, true);
    key_switch_aux->generateDecompAndDigit();
    return *key_switch_aux;
}

RNSPoly& Context::getKeySwitchAux2() {
    if (key_switch_aux2 == nullptr)
        key_switch_aux2 = std::make_unique<RNSPoly>(*this, L, true);
    key_switch_aux2->generateDecompAndDigit();
    return *key_switch_aux2;
}

RNSPoly& Context::getModdownAux() {
    if (moddown_aux == nullptr)
        moddown_aux = std::make_unique<RNSPoly>(*this, L, true);
    return *moddown_aux;
}
std::vector<uint64_t> Context::ElemForEvalMult(int level, const double operand) {

    uint32_t numTowers = level + 1;
    std::vector<lbcrypto::DCRTPoly::Integer> moduli(numTowers);
    for (usint i = 0; i < numTowers; i++) {
        moduli[i] = prime[i].p;
    }

    double scFactor = param.ScalingFactorReal[level];

    typedef int128_t DoubleInteger;
    int32_t MAX_BITS_IN_WORD_LOCAL = 125;

    int32_t logApprox = 0;
    const double res = std::fabs(operand * scFactor);
    if (res > 0) {
        int32_t logSF = static_cast<int32_t>(std::ceil(std::log2(res)));
        int32_t logValid = (logSF <= MAX_BITS_IN_WORD_LOCAL) ? logSF : MAX_BITS_IN_WORD_LOCAL;
        logApprox = logSF - logValid;
    }
    double approxFactor = pow(2, logApprox);

    DoubleInteger large = static_cast<DoubleInteger>(operand / approxFactor * scFactor + 0.5);
    DoubleInteger large_abs = (large < 0 ? -large : large);
    DoubleInteger bound = (uint64_t)1 << 63;

    std::vector<lbcrypto::DCRTPoly::Integer> factors(numTowers);

    if (large_abs > bound) {
        for (usint i = 0; i < numTowers; i++) {
            DoubleInteger reduced = large % moduli[i].ConvertToInt();

            factors[i] = (reduced < 0) ? static_cast<uint64_t>(reduced + moduli[i].ConvertToInt())
                                       : static_cast<uint64_t>(reduced);
        }
    } else {
        int64_t scConstant = static_cast<int64_t>(large);
        for (usint i = 0; i < numTowers; i++) {
            int64_t reduced = scConstant % static_cast<int64_t>(moduli[i].ConvertToInt());

            factors[i] = (reduced < 0) ? reduced + moduli[i].ConvertToInt() : reduced;
        }
    }

    // Scale back up by approxFactor within the CRT multiplications.
    if (logApprox > 0) {
        int32_t logStep = (logApprox <= lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP)
                              ? logApprox
                              : lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP;
        lbcrypto::DCRTPoly::Integer intStep = uint64_t(1) << logStep;
        std::vector<lbcrypto::DCRTPoly::Integer> crtApprox(numTowers, intStep);
        logApprox -= logStep;

        while (logApprox > 0) {
            int32_t logStep = (logApprox <= lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP)
                                  ? logApprox
                                  : lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP;
            lbcrypto::DCRTPoly::Integer intStep = uint64_t(1) << logStep;
            std::vector<lbcrypto::DCRTPoly::Integer> crtSF(numTowers, intStep);
            crtApprox = lbcrypto::CKKSPackedEncoding::CRTMult(crtApprox, crtSF, moduli);
            logApprox -= logStep;
        }
        factors = lbcrypto::CKKSPackedEncoding::CRTMult(factors, crtApprox, moduli);
    }

    std::vector<uint64_t> result(numTowers);
    for (int i = 0; i < result.size(); ++i) {
        result[i] = factors[i].ConvertToInt();
        result[i] = result[i] % prime[i].p;
    }

    return result;
}

std::ostream& operator<<(std::ostream& o, const uint128_t& x) {
    if (x == std::numeric_limits<uint128_t>::min())
        return o << "0";
    if (x < 10)
        return o << (char)(x + '0');
    return o << x / 10 << (char)(x % 10 + '0');
}

std::vector<uint64_t> Context::ElemForEvalAddOrSub(const int level, const double operand, const int noise_deg) {
    usint sizeQl = level + 1;
    std::vector<lbcrypto::DCRTPoly::Integer> moduli(sizeQl);
    for (usint i = 0; i < sizeQl; i++) {
        moduli[i] = prime[i].p;
    }

    //double scFactor = param.ScalingFactorReal.at(level);
    double scFactor = 0;
    if (this->rescaleTechnique == FLEXIBLEAUTOEXT && level == L) {
        scFactor =
            param.ScalingFactorRealBig.at(level);  // cryptoParams->GetScalingFactorRealBig(ciphertext->GetLevel());
    } else {
        scFactor = param.ScalingFactorReal.at(level);  //cryptoParams->GetScalingFactorReal(ciphertext->GetLevel());
    }

    int32_t logApprox = 0;
    const double res = std::fabs(operand * scFactor);
    if (res > 0) {
        int32_t logSF = static_cast<int32_t>(std::ceil(std::log2(res)));
        int32_t logValid = (logSF <= lbcrypto::LargeScalingFactorConstants::MAX_BITS_IN_WORD)
                               ? logSF
                               : lbcrypto::LargeScalingFactorConstants::MAX_BITS_IN_WORD;
        logApprox = logSF - logValid;
    }
    double approxFactor = pow(2, logApprox);

    lbcrypto::DCRTPoly::Integer scConstant = static_cast<uint64_t>(operand * scFactor / approxFactor + 0.5);
    std::vector<lbcrypto::DCRTPoly::Integer> crtConstant(sizeQl, scConstant);

    // Scale back up by approxFactor within the CRT multiplications.
    if (logApprox > 0) {
        int32_t logStep = (logApprox <= lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP)
                              ? logApprox
                              : lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP;
        lbcrypto::DCRTPoly::Integer intStep = uint64_t(1) << logStep;
        std::vector<lbcrypto::DCRTPoly::Integer> crtApprox(sizeQl, intStep);
        logApprox -= logStep;

        while (logApprox > 0) {
            int32_t logStep = (logApprox <= lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP)
                                  ? logApprox
                                  : lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP;
            lbcrypto::DCRTPoly::Integer intStep = uint64_t(1) << logStep;
            std::vector<lbcrypto::DCRTPoly::Integer> crtSF(sizeQl, intStep);
            crtApprox = lbcrypto::CKKSPackedEncoding::CRTMult(crtApprox, crtSF, moduli);
            logApprox -= logStep;
        }
        crtConstant = lbcrypto::CKKSPackedEncoding::CRTMult(crtConstant, crtApprox, moduli);
    }

    // In FLEXIBLEAUTOEXT mode at level 0, we don't use the depth to calculate the scaling factor,
    // so we return the value before taking the depth into account.
    if (this->rescaleTechnique == FLEXIBLEAUTOEXT && level == L) {
        std::vector<uint128_t> result(sizeQl);
        for (int i = 0; i < result.size(); ++i) {
            result[i] = crtConstant[i].ConvertToInt<uint128_t>();
        }

        for (int i = 0; i < result.size(); ++i) {
            result[i] = result[i] % prime[i].p;
        }

        std::vector<uint64_t> result2(crtConstant.size());
        for (int i = 0; i < result.size(); ++i) {
            result2[i] = result[i];
        }

        return result2;
    }

    lbcrypto::DCRTPoly::Integer intScFactor = static_cast<uint64_t>(scFactor + 0.5);
    std::vector<lbcrypto::DCRTPoly::Integer> crtScFactor(sizeQl, intScFactor);

    for (usint i = 1; i < noise_deg; i++) {
        crtConstant = lbcrypto::CKKSPackedEncoding::CRTMult(crtConstant, crtScFactor, moduli);
    }

    std::vector<uint128_t> result(sizeQl);
    for (int i = 0; i < result.size(); ++i) {
        result[i] = crtConstant[i].ConvertToInt<uint128_t>();
    }

    for (int i = 0; i < result.size(); ++i) {
        result[i] = result[i] % prime[i].p;
    }

    std::vector<uint64_t> result2(crtConstant.size());
    for (int i = 0; i < result.size(); ++i) {
        result2[i] = result[i];
    }

    return result2;
}
std::vector<double>& Context::GetCoeffsChebyshev() {
    assert(param.raw != nullptr);
    return param.raw->coefficientsCheby;
}
int Context::GetDoubleAngleIts() {
    assert(param.raw != nullptr);
    return param.raw ? param.raw->doubleAngleIts : 3;
}

int Context::GetBootK() {
    assert(param.raw != nullptr);
    return param.raw ? param.raw->bootK : 1.0;
}

std::map<int, BootstrapPrecomputation> boot_precomps;

BootstrapPrecomputation& Context::GetBootPrecomputation(int slots) {
    if (!boot_precomps.contains(slots))
        assert("No precomputation." == nullptr);
    return boot_precomps[slots];
}

std::map<int, KeySwitchingKey> rot_keys;

KeySwitchingKey& Context::GetRotationKey(int index) {
    //index = index % (cc.N / 2);
    if (index < 0)
        index += this->N / 2;
    return rot_keys.at(index);
}
void Context::AddRotationKey(int index, KeySwitchingKey&& ksk) {
    //index = index % (cc.N / 2);
    if (index < 0)
        index += this->N / 2;
    rot_keys.emplace(index, std::move(ksk));
}
bool Context::HasRotationKey(int index) {
    //index = index % (cc.N / 2);
    if (index < 0)
        index += this->N / 2;
    return rot_keys.contains(index);
}

std::optional<KeySwitchingKey> eval_key;

void Context::AddEvalKey(KeySwitchingKey&& ksk) {
    eval_key.emplace(std::move(ksk));
}
KeySwitchingKey& Context::GetEvalKey() {
    return eval_key.value();
}
Context::~Context() {
    eval_key.reset();
    rot_keys.clear();
    boot_precomps.clear();
}

void Context::AddBootPrecomputation(int slots, BootstrapPrecomputation&& precomp) const {
    {
        std::cout << "Adding bootstrap precomputation to GPU for " << slots << " slots.\n"
                  << "Rotation keys loaded: " << rot_keys.size() << " ~ "
                  << 2 * ((long long)rot_keys.size() * dnum * (L + K + 1) * N * 8 / (1 << 20)) << "MB\n"
                  << "Plaintexts loaded: "
                  << (precomp.CtS.size() == 0 ? (precomp.LT.A.size() + precomp.LT.invA.size())
                                              : (precomp.StC.size() * precomp.StC.at(0).A.size() +
                                                 precomp.CtS.size() * precomp.CtS.at(0).A.size()))
                  << " ~ "
                  << (precomp.CtS.size() == 0
                          ? (precomp.LT.A.size() * precomp.LT.A.at(0).c0.getLevel() +
                             precomp.LT.invA.size() * precomp.LT.invA.at(0).c0.getLevel())
                          : (precomp.StC.size() * precomp.StC.at(0).A.size() * precomp.StC.at(0).A.at(0).c0.getLevel() +
                             precomp.CtS.size() * precomp.CtS.at(0).A.size() *
                                 precomp.CtS.at(0).A.at(0).c0.getLevel())) *
                         N * 8 / (1 << 20)
                  << "MB\n";
    }

    boot_precomps.emplace(slots, std::move(precomp));
}

Context::RESCALE_TECHNIQUE Context::translateRescalingTechnique(lbcrypto::ScalingTechnique technique) {
    return technique == lbcrypto::ScalingTechnique::FIXEDAUTO         ? Context::FIXEDAUTO
           : technique == lbcrypto::ScalingTechnique::FIXEDMANUAL     ? Context::FIXEDMANUAL
           : technique == lbcrypto::ScalingTechnique::FLEXIBLEAUTOEXT ? Context::FLEXIBLEAUTOEXT
           : technique == lbcrypto::ScalingTechnique::FLEXIBLEAUTO    ? Context::FLEXIBLEAUTO
                                                                      : Context::NO_RESCALE;
}

}  // namespace FIDESlib::CKKS
