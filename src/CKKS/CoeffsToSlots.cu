#include "hip/hip_runtime.h"
//
// Created by carlosad on 27/11/24.
//

#include <ranges>
#include <vector>
#include "CKKS/BootstrapPrecomputation.cuh"
#include "CKKS/Ciphertext.cuh"
#include "CKKS/CoeffsToSlots.cuh"
#include "CKKS/Context.cuh"
#include "CKKS/Plaintext.cuh"

using namespace FIDESlib::CKKS;

void FIDESlib::CKKS::EvalLinearTransform(Ciphertext& ctxt, int slots, bool decode) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()});

    /*
    auto pair = m_bootPrecomMap.find(slots);
    if (pair == m_bootPrecomMap.end()) {
        std::string errorMsg(std::string("Precomputations for ") + std::to_string(slots) +
                             std::string(" slots were not generated") +
                             std::string(" Need to call EvalBootstrapSetup and EvalBootstrapKeyGen to proceed"));
        OPENFHE_THROW(errorMsg);
    }
    const std::shared_ptr<CKKSBootstrapPrecom> precom = pair->second;

    auto cc = ct->GetCryptoContext();
    */
    Context& cc = ctxt.cc;
    // Computing the baby-step bStep and the giant-step gStep.
    uint32_t bStep = cc.GetBootPrecomputation(slots).LT.bStep;
    uint32_t gStep = ceil(static_cast<double>(slots) / bStep);

    uint32_t M = cc.N * 2;
    uint32_t N = cc.N;

    // computes the NTTs for each CRT limb (for the hoisted automorphisms used
    // later on)
    //auto digits = cc->EvalFastRotationPrecompute(ct);

    std::vector<Ciphertext> fastRotation;

    for (int i = 0; i < bStep - 1; ++i)
        fastRotation.emplace_back(cc);

    std::vector<Ciphertext*> fastRotationPtr;
    std::vector<int> indexes;
    std::vector<KeySwitchingKey*> keys;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr.push_back(&fastRotation[i - 1]);
        keys.push_back(&cc.GetRotationKey(i));
        indexes.push_back(i);
    }

    if (0) {
        ctxt.rotate_hoisted(keys, indexes, fastRotationPtr);
    } else {
        for (int i = 0; i < bStep - 1; ++i) {
            fastRotation[i].rotate(ctxt, i + 1, cc.GetRotationKey(i + 1));
            //hipDeviceSynchronize();
        }
        hipDeviceSynchronize();
    }
    Ciphertext result(cc);
    Ciphertext inner(cc);
    std::vector<Plaintext>& A = decode ? cc.GetBootPrecomputation(slots).LT.invA : cc.GetBootPrecomputation(slots).LT.A;

    for (uint32_t j = 0; j < gStep; j++) {

        inner.multPt(ctxt, A[bStep * j], false);
        for (uint32_t i = 1; i < bStep; i++) {
            if (bStep * j + i < slots) {
                inner.addMultPt(fastRotation[i - 1], A[bStep * j + i], false);
            }
        }
        // Does rotate -> rescale work???
        //inner.rescale();
        if (j == 0) {
            result.copy(inner);
        } else {
            inner.rotate(bStep * j, cc.GetRotationKey(bStep * j));
            result.add(inner);
        }
    }

    ctxt.copy(result);
    hipDeviceSynchronize();
}

void FIDESlib::CKKS::EvalCoeffsToSlots(Ciphertext& ctxt, int slots, bool decode) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()});

    Context& cc = ctxt.cc;
    //  No need for Encrypted Bit Reverse
    Ciphertext& result = ctxt;
    // hoisted automorphisms
    if (result.NoiseLevel == 2)
        result.rescale();
    std::vector<Ciphertext> auxiliar;

    Ciphertext outer(cc);
    Ciphertext inner(cc);

    int steps = 0;
    for (BootstrapPrecomputation::LTstep& step :
         (decode ? cc.GetBootPrecomputation(slots).StC : cc.GetBootPrecomputation(slots).CtS)) {
        // computes the NTTs for each CRT limb (for the hoisted automorphisms used later on)

        std::vector<Ciphertext*> fastRotationPtr;
        std::vector<int> indexes;
        std::vector<KeySwitchingKey*> keys;

        for (int i = 0; i < step.bStep; ++i) {
            if (i >= auxiliar.size()) {
                auxiliar.emplace_back(cc);
            }
        }
        for (int i = 0; i < step.bStep; ++i) {
            fastRotationPtr.push_back(&auxiliar[i]);
            keys.push_back(step.rotIn[i] ? &cc.GetRotationKey(step.rotIn[i]) : nullptr);
            indexes.push_back(step.rotIn[i]);
        }

        result.rotate_hoisted(keys, indexes, fastRotationPtr);
        for (int32_t i = 0; i < step.gStep; i++) {

            // for the first iteration with j=0:
            int32_t G = step.bStep * i;
            inner.multPt(auxiliar[0], step.A[G], false);
            // continue the loop
            for (int32_t j = 1; j < step.bStep; j++) {
                if ((G + j) != step.slots) {
                    inner.addMultPt(auxiliar[j], step.A[G + j], false);
                }
            }

            if (i == 0) {
                outer.copy(inner);
            } else {
                if (step.rotOut[i] != 0) {
                    inner.rotate(step.rotOut[i], cc.GetRotationKey(step.rotOut[i]));
                }
                outer.add(inner);
            }
        }

        steps++;
        if (steps != (decode ? cc.GetBootPrecomputation(slots).StC : cc.GetBootPrecomputation(slots).CtS).size())
            outer.rescale();
        result.copy(outer);
    }

    CudaCheckErrorMod;
}
