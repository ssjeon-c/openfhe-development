#include "hip/hip_runtime.h"
//
// Created by carlosad on 4/12/24.
//

#include "CKKS/ApproxModEval.cuh"
#include "CKKS/Bootstrap.cuh"
#include "CKKS/BootstrapPrecomputation.cuh"
#include "CKKS/Ciphertext.cuh"
#include "CKKS/CoeffsToSlots.cuh"
#include "CKKS/Context.cuh"

using namespace FIDESlib::CKKS;

constexpr bool PRINT = false;

void FIDESlib::CKKS::Bootstrap(Ciphertext& ctxt, const int slots) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()});

    Context& cc = ctxt.cc;
    /////////////////////////////////////////////////////////////////////
    //NativeInteger q = elementParamsRaisedPtr->GetParams()[0]->GetModulus().ConvertToInt();
    uint64_t q = cc.prime[0].p;
    double qDouble = (double)q;  //q.ConvertToDouble();

    if constexpr (PRINT) {
        std::cout << "q: " << q << " ";
        std::cout << qDouble << std::endl;
    }
    const auto p = cc.param.raw->p;  //cryptoParams->GetPlaintextModulus();
    double powP = pow(2, p);

    if constexpr (PRINT) {
        std::cout << "p: " << p << std::endl;
    }
    int32_t deg = std::round(std::log2(qDouble / powP));
    /*
#if NATIVEINT != 128
    if (deg > static_cast<int32_t>(m_correctionFactor)) {
        OPENFHE_THROW("Degree [" + std::to_string(deg) + "] must be less than or equal to the correction factor [" +
                      std::to_string(m_correctionFactor) + "].");
    }
#endif
    */
    uint32_t correction = cc.GetBootPrecomputation(slots).correctionFactor - deg;
    if constexpr (PRINT)
        std::cout << cc.GetBootPrecomputation(slots).correctionFactor << " " << deg << std::endl;
    double post = std::pow(2, static_cast<double>(deg));

    double pre = 1. / post;
    uint64_t scalar = std::llround(post);

    //////////////////////////////////////////////////////////////////////

    //------------------------------------------------------------------------------
    // RAISING THE MODULUS
    //------------------------------------------------------------------------------

    // In FLEXIBLEAUTO, raising the ciphertext to a larger number
    // of towers is a bit more complex, because we need to adjust
    // it's scaling factor to the one that corresponds to the level
    // it's being raised to.
    // Increasing the modulus

    if (ctxt.NoiseLevel == 2)
        ctxt.rescale();
    if constexpr (PRINT) {
        std::cout << "Initial ";
        for (auto& j : ctxt.c0.GPU) {
            for (auto& i : j.limb)
                SWITCH(i, printThisLimb(1));
        }
        std::cout << std::endl;
        std::cout << correction << std::endl;
        std::cout << std::pow((double)2.0, (double)-1.0 * (double)correction) << std::endl;
    }

    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        uint32_t lvl = cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT;
        double targetSF = cc.param.ScalingFactorReal[cc.L - lvl];
        double sourceSF = ctxt.NoiseFactor;        // ciphertext->GetScalingFactor();
        uint32_t numTowers = ctxt.getLevel() + 1;  // ciphertext->GetElements()[0].GetNumOfElements();
        double modToDrop = static_cast<double>(cc.prime.at(numTowers - 1).p);
        //cryptoParams->GetElementParams()->GetParams()[numTowers - 1]->GetModulus().ConvertToDouble();

        // in the case of FLEXIBLEAUTO, we need to bring the ciphertext to the right scale using a
        // a scaling multiplication. Note the at currently FLEXIBLEAUTO is only supported for NATIVEINT = 64.
        // So the other branch is for future purposes (in case we decide to add add the FLEXIBLEAUTO support
        // for NATIVEINT = 128.
        // Scaling down the message by a correction factor to emulate using a larger q0.
        // This step is needed so we could use a scaling factor of up to 2^59 with q9 ~= 2^60.
        double adjustmentFactor = (targetSF / sourceSF) * (modToDrop / sourceSF);
        double pow = std::pow((double)2.0, (double)-1.0 * (double)correction);
        adjustmentFactor *= pow;
        if constexpr (PRINT)
            std::cout << adjustmentFactor << std::endl;

        ctxt.multScalar(adjustmentFactor);
        //cc->EvalMultInPlace(ciphertext, adjustmentFactor);
        ctxt.rescale();
        //algo->ModReduceInternalInPlace(ciphertext, BASE_NUM_LEVELS_TO_DROP);
        ctxt.NoiseFactor = targetSF;
        //ciphertext->SetScalingFactor(targetSF);
    } else {  // THIS is only for FIXEDAUTO/FIXEDMANUAL (AdjustCiphertext)
        // Scaling down the message by a correction factor to emulate using a larger q0.
        // This step is needed so we could use a scaling factor of up to 2^59 with q9 ~= 2^60.
        ctxt.multScalar(std::pow((double)2.0, (double)-1.0 * (double)correction), false);
        ctxt.rescale();
    }
    // auto ctxtDCRT = raised->GetElements();
    if constexpr (PRINT) {
        std::cout << "Adjustment ";
        for (auto& j : ctxt.c0.GPU)
            for (auto& i : j.limb) {
                SWITCH(i, printThisLimb(1));
            }
        std::cout << std::endl;
    }

    ctxt.c0.INTT(cc.batch);
    if constexpr (PRINT) {
        std::cout << "Adjustment ";
        for (auto& j : ctxt.c0.GPU)
            for (auto& i : j.limb) {
                SWITCH(i, printThisLimb(1));
            }
        std::cout << std::endl;
    }
    hipDeviceSynchronize();
    ctxt.c0.grow(cc.L - (cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT), false);
    hipDeviceSynchronize();
    ctxt.c0.broadcastLimb0();
    hipDeviceSynchronize();
    if constexpr (PRINT) {
        std::cout << "Adjustment ";
        for (auto& j : ctxt.c0.GPU)
            for (auto& i : j.limb) {
                SWITCH(i, printThisLimb(1));
            }
        std::cout << std::endl;
    }
    ctxt.c0.NTT(cc.batch);
    ctxt.c1.INTT(cc.batch);
    hipDeviceSynchronize();
    ctxt.c1.grow(cc.L - (cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT), false);
    hipDeviceSynchronize();
    ctxt.c1.broadcastLimb0();
    hipDeviceSynchronize();
    ctxt.c1.NTT(cc.batch);
    std::cout << ctxt.c0.getLevel()  << " jaslkdjaslkdj \n";
    if constexpr (PRINT) {
        std::cout << "ModRaise ";
        for (auto& j : ctxt.c0.GPU)
            for (auto& i : j.limb) {
                SWITCH(i, printThisLimb(1));
            }
        std::cout << std::endl;
    }
    //------------------------------------------------------------------------------
    // SETTING PARAMETERS FOR APPROXIMATE MODULAR REDUCTION
    //------------------------------------------------------------------------------

    // Coefficients of the Chebyshev series interpolating 1/(2 Pi) Sin(2 Pi K x)
    double k = cc.GetBootK();

    double constantEvalMult = pre * (1.0 / (k * cc.N));

    if constexpr (PRINT)
        std::cout << "mult: " << constantEvalMult << std::endl;
    CudaCheckErrorMod;
    ctxt.multScalar(constantEvalMult, false);

    if constexpr (PRINT) {
        std::cout << "Raise scaled ";
        for (auto& j : ctxt.c0.GPU)
            for (auto& i : j.limb) {
                SWITCH(i, printThisLimb(1));
            }
        std::cout << std::endl;
    }

    ////////////////////////////////////////////////////////////////

    bool isLT = cc.GetBootPrecomputation(slots).LT.slots == slots;
    Ciphertext aux(cc);

    if (cc.N / 2 != slots) {
        for (int j = 1; j < cc.N / (2 * slots); j <<= 1) {
            aux.rotate(ctxt, j * slots, cc.GetRotationKey(j * slots));
            ctxt.add(aux);
        }
    }
    if (ctxt.NoiseLevel == 2) {
        ctxt.rescale();
    }

    if (isLT) {
        hipDeviceSynchronize();
        EvalLinearTransform(ctxt, slots, false);
        hipDeviceSynchronize();
    } else
        EvalCoeffsToSlots(ctxt, slots, false);

    if (cc.N / 2 == slots) {
        hipDeviceSynchronize();
        aux.conjugate(ctxt);
        hipDeviceSynchronize();
        Ciphertext ctxtEncI(cc);
        hipDeviceSynchronize();
        ctxtEncI.sub(ctxt, aux);
        hipDeviceSynchronize();
        ctxt.add(aux);
        hipDeviceSynchronize();
        multMonomial(ctxtEncI, 3 * 2 * cc.N / 4);
        hipDeviceSynchronize();
        //ctxt.copy(ctxtEncI);
        hipDeviceSynchronize();
        ctxt.rescale();
        ctxtEncI.rescale();
        approxModReduction(ctxt, ctxtEncI, cc.GetEvalKey(), scalar);
        hipDeviceSynchronize();
    } else {
        /*
        if (ctxt.NoiseLevel == 2) {

        }
*/
        hipDeviceSynchronize();
        aux.conjugate(ctxt);
        hipDeviceSynchronize();
        ctxt.add(aux);
        hipDeviceSynchronize();

        if (cc.rescaleTechnique == Context::FIXEDMANUAL)
            ctxt.rescale();

        approxModReductionSparse(ctxt, cc.GetEvalKey(), scalar);
        if constexpr (PRINT)
            std::cout << "Scalar last " << scalar << std::endl;
        hipDeviceSynchronize();
    }

    if (ctxt.NoiseLevel == 2) {
        ctxt.rescale();
    }

    if (isLT)
        EvalLinearTransform(ctxt, slots, true);
    else
        EvalCoeffsToSlots(ctxt, slots, true);

    if (cc.N / 2 != slots) {
        aux.rotate(ctxt, slots, cc.GetRotationKey(slots));
        ctxt.add(aux);
    }

    uint64_t corFactor = (uint64_t)1 << std::llround(correction);
    multIntScalar(ctxt, corFactor);
    if constexpr (PRINT) {
        for (auto& j : ctxt.c0.GPU)
            for (auto& i : j.limb) {
                SWITCH(i, printThisLimb(2));
            }
        std::cout << std::endl;
    }
}
